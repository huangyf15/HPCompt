#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <Windows.h>
#include <io.h>
#include <string>
#include <math.h>
#include <time.h>
#include <chrono>

using namespace std;
using namespace chrono;

#define DEBUGMODE 1
#define RUNMODE 0
#define PRECISION 1e-5

// totalWidth/BLOCK_WIDTH should be integer
// -- shared memory is used and no elsecase
#define BLOCK_WIDTH 8
int totalWidth = 2048;

void random_ints(int* a, int n){ for (int i = 0; i < n; ++i)	a[i] = rand()%10;}

string getCurrTimeStr();

void mulMatrixOnHost(int* M, int* N, int* P, int totalWidth);

inline int map2MatrixEleNo(int rowNo, int colNo, int height) { return rowNo * height + colNo; }

void printResMatrix(string info, float seconds, int* mat, int totalWidth);

// can only deal with matrix size < 32 * 32
__global__ void mulMatrixKernel_singleBlock(int* d_M, int* d_N, int* d_P, int totalWidth)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;

	for (int k = 0; k < totalWidth; ++k)
	{
		int d_Mele = d_M[ty * totalWidth + k];
		int d_Nele = d_N[k * totalWidth + tx];
		Pvalue += d_Mele * d_Nele;
	}
	d_P[ty * totalWidth + tx] = Pvalue;
}

__global__ void mulMatrixKernel_globalMem(int* d_M, int* d_N, int* d_P, int totalWidth)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int Pvalue = 0;

	for (int k = 0; k < totalWidth; ++k)
	{
		Pvalue += d_M[row * totalWidth + k] * d_N[k * totalWidth + col];
	}

	d_P[row * totalWidth + col] = Pvalue;
}

__global__ void mulMatrixKernel_sharedMem(int* d_M, int* d_N, int* d_P, int totalWidth)
{
  __shared__ int d_Ms[BLOCK_WIDTH][BLOCK_WIDTH];
  __shared__ int d_Ns[BLOCK_WIDTH][BLOCK_WIDTH];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
	int row = by * BLOCK_WIDTH + ty;
	int col = bx * BLOCK_WIDTH + tx;

	int Pvalue = 0;

  for (int m = 0; m < totalWidth/BLOCK_WIDTH; ++m)
  {
    d_Ms[ty][tx] = d_M[row*totalWidth + (m*BLOCK_WIDTH + tx)];
    d_Ns[ty][tx] = d_N[col + (m*BLOCK_WIDTH + ty)*totalWidth];
    __syncthreads();
    for (int k = 0; k < BLOCK_WIDTH; ++k)
    {
      Pvalue += d_Ms[ty][k] * d_Ns[k][tx];
      __syncthreads();
    }
  }

	d_P[row * totalWidth + col] = Pvalue;
}

int main(int argc, char** argv)
{
  int thisTest;
  if (argc > 1) 
  {
    thisTest = atoi(argv[1]);
  }
  else
  {
    thisTest = 1;
  }

	static string testName = "shared memory effect on matMulti_CUDA";
  string folderPath = "res_matMulti_CUDA_gs";
	static string opFileName = folderPath+"/out_gs_w"+to_string(totalWidth)\
                                   +"_b"+to_string(BLOCK_WIDTH)+".log";

  if (~_access(folderPath.c_str(), 0))
  {
    string command;
    command = "mkdir " + folderPath;  
    system(command.c_str());
  }
	ofstream fout(opFileName);
	streambuf* oldclog;
	oldclog = clog.rdbuf(fout.rdbuf());

	clog << "Title: " << testName << "\n"
		<< "Current time: " << getCurrTimeStr() << " ms\n\n"
		<< "Init matrices: Width = " << totalWidth << "\n"
		<< "CUDA blocksize = " << BLOCK_WIDTH << "\n" << endl;

	int totalEleNum = totalWidth * totalWidth;
	int* matA, * matB, * matC_g, * matC_s, * matCRef;
	int size_of_matrix = totalEleNum * sizeof(int);

	matA = (int*)malloc(size_of_matrix); random_ints(matA, totalEleNum);
	matB = (int*)malloc(size_of_matrix); random_ints(matB, totalEleNum);
	matC_g = (int*)malloc(size_of_matrix); random_ints(matC_g, totalEleNum);
	matC_s = (int*)malloc(size_of_matrix); random_ints(matC_s, totalEleNum);
	matCRef = (int*)malloc(size_of_matrix); random_ints(matCRef, totalEleNum);

	// CPU compt
	auto start = system_clock::now();
	mulMatrixOnHost(matA, matB, matCRef, totalWidth);
	auto end = system_clock::now();
	auto duration = duration_cast<microseconds>(end - start);
	float seconds = float(duration.count()) * microseconds::period::num \
		/ microseconds::period::den;
  clog << "--hostFunc() elapsed " << seconds << " s..\n\n";

	// GPU compt: preparation
	int* d_matA, * d_matB, * d_matC_g, * d_matC_s;

	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 dimGrid((totalWidth + dimBlock.x - 1) / dimBlock.x, \
               (totalWidth + dimBlock.y - 1) / dimBlock.y);

	hipMalloc((void**)&d_matA, size_of_matrix);
	hipMalloc((void**)&d_matB, size_of_matrix);

	hipMemcpy(d_matA, matA, size_of_matrix, hipMemcpyHostToDevice);
	hipMemcpy(d_matB, matB, size_of_matrix, hipMemcpyHostToDevice);

  // use global memory
	start = system_clock::now();

	hipMalloc((void**)&d_matC_g, size_of_matrix);
	mulMatrixKernel_globalMem << <dimGrid, dimBlock >> > (d_matA, d_matB, d_matC_g, totalWidth);
	hipMemcpy(matC_g, d_matC_g, size_of_matrix, hipMemcpyDeviceToHost);

	end = system_clock::now();
	duration = duration_cast<microseconds>(end - start);
	seconds = float(duration.count()) * microseconds::period::num \
		/ microseconds::period::den;
  clog << "--GPU_gFunc() elapsed " << seconds << " s..\n\n";

  // use shared memory
	start = system_clock::now();

	hipMalloc((void**)&d_matC_s, size_of_matrix);
	mulMatrixKernel_sharedMem << <dimGrid, dimBlock >> > (d_matA, d_matB, d_matC_s, totalWidth);
	hipMemcpy(matC_s, d_matC_s, size_of_matrix, hipMemcpyDeviceToHost);

	end = system_clock::now();
	duration = duration_cast<microseconds>(end - start);
	seconds = float(duration.count()) * microseconds::period::num \
		/ microseconds::period::den;
  clog << "--GPU_sFunc() elapsed " << seconds << " s..\n\n";

  // check device results
  for (int i = 0; i < totalEleNum; ++i)
  {
    if (fabs(matCRef[i] - matC_g[i]) > PRECISION)
    {
      fprintf(stderr,"Result verification failed (GPU_g) at element %d\n",i);
      exit(EXIT_FAILURE);
    }
    else if (fabs(matCRef[i] - matC_s[i]) > PRECISION)
    {
      fprintf(stderr,"Result verification failed (GPU_s) at element %d\n",i);
      exit(EXIT_FAILURE);
    }
  }

  // free device global memory
	hipFree(d_matA);
	hipFree(d_matB);
	hipFree(d_matC_g);
	hipFree(d_matC_s);

  // free host memory
	free(matA);
	free(matB);
	free(matC_g);
	free(matC_s);
	free(matCRef);

  return 0;
}

string getCurrTimeStr()
{
	system_clock::time_point t = system_clock::now();
	milliseconds ms = duration_cast<milliseconds>(t.time_since_epoch());
	char time_string[128];
	time_t curtm = time(NULL);
	struct tm tm;
	localtime_s(&tm, &curtm);
	sprintf_s(time_string, "%04d-%02d-%02d %02d:%02d:%02d %03lld ", \
		tm.tm_year + 1900, tm.tm_mon + 1, tm.tm_mday, tm.tm_hour, \
		tm.tm_min, tm.tm_sec, ms.count() % 1000);
	return time_string;
}

void mulMatrixOnHost(int* M, int* N, int* P, int totalWidth)
{
	for (int i = 0; i < totalWidth; ++i)
	{
		for (int j = 0; j < totalWidth; ++j)
		{
			int sum = 0;
			for (int k = 0; k < totalWidth; ++k)
			{
				sum += M[map2MatrixEleNo(i, k, totalWidth)] * \
					N[map2MatrixEleNo(k, j, totalWidth)];
			}
			P[map2MatrixEleNo(i, j, totalWidth)] = sum;
		}
	}
}

void printResMatrix(string info, float seconds, int* mat, int totalWidth)
{
	clog << "*** Using " << info << ": cost "
		<< seconds << " s" << endl;
	int outputWidth = min(totalWidth, 10);
	for (int i = 0; i < outputWidth; i++)
	{
		for (int j = 0; j < outputWidth; j++)
		{
			clog << mat[map2MatrixEleNo(i, j, totalWidth)] << "\t";
		}
		clog << endl << endl;
	}
}